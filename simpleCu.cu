#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "simpleMPI.h"

__global__ void simpleMPIKernel(float *input, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = pow(input[tid],2);
}


void initData(float *data, int dataSize)
{
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = (float)(rand()%10+1);
    }
}


void printData(float *data, int dataSize)
{
    for (int i = 0; i < dataSize; i++)
    {
        printf("%10f",data[i]);
    }
	printf("\n");
}


void computeGPU(float *hostData, int blockSize, int gridSize)
{
    int dataSize = blockSize * gridSize;

    float *deviceInputData = NULL;
    hipMalloc((void **)&deviceInputData, dataSize * sizeof(float));

    float *deviceOutputData = NULL;
    hipMalloc((void **)&deviceOutputData, dataSize * sizeof(float));

   hipMemcpy(deviceInputData, hostData, dataSize * sizeof(float), hipMemcpyHostToDevice);

    simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputData, deviceOutputData);

    hipMemcpy(hostData, deviceOutputData, dataSize *sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceInputData);
    hipFree(deviceOutputData);
}


